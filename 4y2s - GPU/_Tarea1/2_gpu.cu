
#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 1024
#define BLOCK_SIZE 16


// global, es llamada por el host, pero ejecutada en gpu
// estas deben ser tipo void
// variaciones incluyen
/*
    __device__; es llamada en device, solo puede ser llamada dentro de un host u device
    __host__; el prefijo por default, llamada y ejecutada en el CPU (host)
*/
__global__ void matrixAdd(int *A, int *B, int *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < SIZE && col < SIZE) {
        C[row * SIZE + col] = A[row * SIZE + col] + B[row * SIZE + col];
    }
}

int main() {
    
    // size es una cantidad de bytes
    // SIZE es el tamaño de las matrices
    int size = SIZE * SIZE * sizeof(int);
    
    
    // Reservar memoria en el host
    int *A, *B, *C;
    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);

    // Inicializar las matrices A y B, de manera plana
    for(int i = 0; i < SIZE * SIZE; i++) {
        A[i] = 1;
        B[i] = 2;
    }

    // Reservar memoria en el device
    int *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copiar las matrices A y B al device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Definir la configuración de ejecución
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((SIZE + dimBlock.x - 1) / dimBlock.x, (SIZE + dimBlock.y - 1) / dimBlock.y);

    // Lanzar el kernel
    // el kernel se lanza con 
    // kernelfuncion<<<dimGrid,dimBlock>>>(parametros**);
    matrixAdd<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Copiar la matriz C del device al host
    // esto implica que movamos lo realizado en GPU (device) devuelta al sistema principal (host)
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    /* ----------- Verificacion de Resultados ---------------------*/
    // Verificar que todos los elementos de C son iguales a 3
    for(int i = 0; i < SIZE * SIZE; i++) {
        if(C[i] != 3) {
            printf("Error: el elemento C[%d] no es igual a 3\n", i);
            return 1;
        }
    }

    printf("Todos los elementos de la matriz C son iguales a 3\n");

    // Liberar memoria
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
