
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define SIZE 1024
#define BLOCK_SIZE 16
#define NUM_STREAMS 4

#define CUDA_CHECK_ERROR() __cuda_check_errors(__FILE__, __LINE__)
#define CUDA_SAFE_CALL(err) __cuda_safe_call(err, __FILE__, __LINE__)

inline void __cuda_check_errors (const char *filename, const int line_number);
inline void __cuda_safe_call (hipError_t err, const char *filename, const int line_number);



__global__ void matrixAddGlobal(int *A, int *B, int *C, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;


    if (row < SIZE && col < SIZE) {
        C[row * SIZE + col] = A[row * SIZE + col] + B[row * SIZE + col];
    }
}

int main() {
    clock_t start, end;
    double cpu_time_used, total_time = 0;

    printf("Todos los elementos de la matriz C son iguales a 3\n");

    // size es una cantidad de bytes
    // SIZE es el tamaño de las matrices
    int size = SIZE * SIZE * sizeof(int);

    start = clock();
    // Reservar memoria en el host
    int *A, *B, *C;
    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);

    // Inicializar las matrices A y B, de manera plana
    for (int i = 0; i < SIZE * SIZE; i++) {
        A[i] = 1;
        B[i] = 2;
    }
    end = clock();
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    total_time += cpu_time_used;
    printf("\t asignación A,B,C en %f ms\n", total_time * 1000);

    // ---------- device ................................
    // Reservar memoria en el device
    int *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copiar las matrices A y B al device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Definir la configuración de ejecución
    // dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    // dim3 dimGrid((SIZE + dimBlock.x - 1) / dimBlock.x, (SIZE + dimBlock.y - 1) / dimBlock.y);
    // printf("\t ----dimGrid.x: %d, dimGrid.y: %d\n", dimGrid.x, dimGrid.y);
    // printf("\t ----dimBlock.x: %d, dimBlock.y: %d\n", dimBlock.x, dimBlock.y);

    // Crear streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    int streamSize = SIZE / NUM_STREAMS;
    int streamBytes = streamSize * SIZE * sizeof(int);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((SIZE + dimBlock.x - 1) / dimBlock.x, (SIZE + dimBlock.y - 1) / dimBlock.y);
    printf("\t ----dimGrid.x: %d, dimGrid.y: %d\n", dimGrid.x, dimGrid.y);
    printf("\t ----dimBlock.x: %d, dimBlock.y: %d\n", dimBlock.x, dimBlock.y);

    start = clock();
    // Distribuir el cálculo entre los streams
    
    for (int i = 0; i < NUM_STREAMS; i++) {
        // nota para futuro:
        // !!!!!!!
        // no es necesario especificar offset, pues ya se hace con threadId
        /*
        cudaMemcpyAsync(d_A + offset* SIZE, A + offset * SIZE, streamBytes, cudaMemcpyHostToDevice, streams[i]);
        cudaMemcpyAsync(d_B + offset * SIZE, B + offset * SIZE, streamBytes, cudaMemcpyHostToDevice, streams[i]);
        matrixAddGlobal<<<dimGrid, dimBlock, 0, streams[i]>>>(d_A + offset * SIZE, d_B + offset * SIZE, d_C + offset * SIZE, SIZE);
        */

        // Copiar porciones de las matrices A y B a los streams
        hipMemcpyAsync(d_A, A, streamBytes, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_B, B, streamBytes, hipMemcpyHostToDevice, streams[i]);

        // Lanzar el kernel en cada stream
        matrixAddGlobal<<<dimGrid, dimBlock, 0, streams[i]>>>(d_A, d_B, d_C, SIZE);
    }

    // Sincronizar los streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        //cudaStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }



    end = clock();
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    total_time += cpu_time_used;
    printf("\t kernel en: %f ms\n", total_time * 1000);

    // Copiar la matriz C del device al host
    start = clock();
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    end = clock();
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    total_time += cpu_time_used;
    printf("\t copia a host en: %f ms\n", total_time * 1000);




    /* ----------- Verificación de Resultados ---------------------*/
    // Verificar que todos los elementos de C son iguales a 3
    for (int i = 0; i < SIZE * SIZE; i++) {
        if (C[i] != 3) {
            printf("C[0]=%d \n",C[0]);
            printf("Error: el elemento C[%d]=%d no es igual a 3\n", i, C[i]);
            free(A);
            free(B);
            free(C);
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C);
            return 1;
        }
    }

    printf("Todos los elementos de la matriz C son iguales a 3\n");

    // Liberar memoria
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    printf("Limpiando punteros en device y host\n");
    printf("Tiempo total %f\n", total_time);
    return 0;
}

/* Encontre codigo para intentar arreglar mi problema de C[256]=0*/

// See: http://codeyarns.com/2011/03/02/how-to-do-error-checking-in-cuda/
inline void
__cuda_check_errors (const char *filename, const int line_number)
{
  hipError_t err = hipDeviceSynchronize ();
  if (err != hipSuccess)
    {
      printf ("CUDA error %i at %s:%i: %s\n",
          err, filename, line_number, hipGetErrorString (err));
      exit (-1);
    }
}

inline void
__cuda_safe_call (hipError_t err, const char *filename, const int line_number)
{
  if (err != hipSuccess)
    {
      printf ("CUDA error %i at %s:%i: %s\n",
          err, filename, line_number, hipGetErrorString (err));
      exit (-1);
    }
}