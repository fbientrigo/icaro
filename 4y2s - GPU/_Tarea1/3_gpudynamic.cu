
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define SIZE 1024
#define BLOCK_SIZE 16


__global__ void matrixAddShared(int *A, int *B, int *C) {
    /*se reserva memoria compartida dentro del kernel*/
    extern __shared__ int shared_memory[];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Copiar datos de la matriz A a la memoria compartida
    shared_memory[threadIdx.y * blockDim.x + threadIdx.x] = A[row * SIZE + col];
    /*cada lugar en la memoria compartida contiene la información de A*/

    // todos tienen acceso a la memoria compartida
    // por tanto B, deberia de encontrarse más "contiguo" a esta memoria que previamente a A

    __syncthreads();

    // Realizar la suma utilizando la memoria compartida
    // cada grid y block de B contiene memoria compartida
    if(row < SIZE && col < SIZE) {
        C[row * SIZE + col] = shared_memory[threadIdx.y * blockDim.x + threadIdx.x] + B[row * SIZE + col];
    }
}


int main() {
    clock_t start, end;
    double cpu_time_used, total_time = 0;

    printf("Todos los elementos de la matriz C son iguales a 3\n");


    // size es una cantidad de bytes
    // SIZE es el tamaño de las matrices
    int size = SIZE * SIZE * sizeof(int);
    


    start = clock();
    // Reservar memoria en el host
    int *A, *B, *C;
    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);

    // Inicializar las matrices A y B, de manera plana
    for(int i = 0; i < SIZE * SIZE; i++) {
        A[i] = 1;
        B[i] = 2;
    }
    end = clock(); cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC; total_time += cpu_time_used;
    printf("\t asignación A,B,C en %f ms\n", total_time * 1000);


    // ---------- device ................................
    // reservar memoria compartida
    start = clock();
    // Reservar memoria en el device
    int *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copiar las matrices A y B al device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Definir la configuración de ejecución
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((SIZE + dimBlock.x - 1) / dimBlock.x, (SIZE + dimBlock.y - 1) / dimBlock.y);
    printf("\t ----dimGrid.x: %d, dimGrid.y: %d\n", dimGrid.x, dimGrid.y);
    printf("\t ----dimBlock.x: %d, dimBlock.y: %d\n", dimBlock.x, dimBlock.y);

    end = clock(); cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC; total_time += cpu_time_used;
    printf("\t asignación en Device en: %f ms\n", total_time * 1000);

    // Lanzar el kernel
    start = clock();
    // el kernel se lanza con 
    // kernelfuncion<<<dimGrid,dimBlock, sharedMem>>>(parametros**);


    // Tamaño de la memoria compartida por bloque
    int shared_memory_size = BLOCK_SIZE * BLOCK_SIZE * sizeof(int);
    matrixAddShared<<<dimGrid, dimBlock, shared_memory_size>>>(d_A, d_B, d_C);
    
    end = clock(); cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC; total_time += cpu_time_used;
    printf("\t kernel en: %f ms\n", total_time * 1000);
    //matrixAdd<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Copiar la matriz C del device al host
    start = clock();
    // esto implica que movamos lo realizado en GPU (device) devuelta al sistema principal (host)
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    end = clock(); cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC; total_time += cpu_time_used;
    printf("\t copia a host en: %f ms\n", total_time * 1000);

    /* ----------- Verificacion de Resultados ---------------------*/
    // Verificar que todos los elementos de C son iguales a 3
    for(int i = 0; i < SIZE * SIZE; i++) {
        if(C[i] != 3) {
            printf("Error: el elemento C[%d] no es igual a 3\n", i);
            free(A);        free(B);         free(C);
            hipFree(d_A);  hipFree(d_B);   hipFree(d_C);
            return 1;
        }
    }

    printf("Todos los elementos de la matriz C son iguales a 3\n");

    // Liberar memoria
    free(A);        free(B);         free(C);
    hipFree(d_A);  hipFree(d_B);   hipFree(d_C);
    printf("Limpiando punteros en device y host\n");
    return 0;
}
